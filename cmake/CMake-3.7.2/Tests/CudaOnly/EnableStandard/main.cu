

#include <hip/hip_runtime.h>
#include <iostream>

int static_cuda11_func(int);
int shared_cuda11_func(int);

void test_functions()
{
  static_cuda11_func( int(42) );
  shared_cuda11_func( int(42) );
}

int main(int argc, char **argv)
{
  test_functions();
  return 0;
}
