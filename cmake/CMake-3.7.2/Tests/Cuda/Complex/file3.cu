#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

static
__global__
void file3_kernel(result_type& r, int x)
{
  r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

int file3_launch_kernel(int x)
{
  result_type r;
  file3_kernel <<<1,1>>> (r,x);
  hipError_t err = hipGetLastError();
  if(err == hipSuccess)
    {
    std::cerr << hipGetErrorString(err) << std::endl;
    return x;
    }
  return r.sum;
}
