#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

void __host__ cuda_dynamic_lib_func();

static
__global__
void mixed_kernel(result_type& r, int x)
{
  r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

int mixed_launch_kernel(int x)
{
  cuda_dynamic_lib_func();

  result_type r;
  mixed_kernel <<<1,1>>> (r,x);
  return r.sum;
}
