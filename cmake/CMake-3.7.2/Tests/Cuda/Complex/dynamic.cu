
#include <string>
#include <hip/hip_runtime.h>
#include <iostream>

int dynamic_base_func(int);

int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static
__global__
void DetermineIfValidCudaDevice()
{
}

void cuda_dynamic_lib_func()
{
  DetermineIfValidCudaDevice <<<1,1>>> ();
  hipError_t err = hipGetLastError();
  if(err == hipSuccess)
    {
    std::cerr << hipGetErrorString(err) << std::endl;
    }
}
